#include "hip/hip_runtime.h"
#include <iostream>
#include<thread>
#include "hip/hip_runtime.h"
#include "../header/gpu.hpp"

#include<opencv2/core/core.hpp>
#include<opencv2/imgcodecs.hpp>
#include<opencv2/highgui/highgui.hpp>

using namespace cv;


using namespace std;

__global__ void kernel0()
{
	printf("hello world! -> 0\n");
}

__global__ void kernel1()
{
	printf("hello world! -> 1\n");
}

__global__ void kernel2()
{
	printf("hello world! -> 2\n");
}

__global__ void kernel3()
{
	printf("hello world! -> 3\n");
}



void f1(){
	int err = hipSetDevice(0);
        printf("set device 0  %d \n", err);
	kernel0 << <1, 1 >> >();
        hipDeviceSynchronize();
}

void f2(){
	int err = hipSetDevice(1);
	printf("set device 1  %d \n", err);
	kernel1 << <1, 1 >> >();
        hipDeviceSynchronize();
}

void f3(){
	int err = hipSetDevice(2);
	printf("set device 2  %d \n", err);
	kernel2 << <1, 1 >> >();
        hipDeviceSynchronize();
}

void f4(){
	int err = hipSetDevice(3);
	printf("set device 3  %d \n", err);
	kernel3 << <1, 1 >> >();
        hipDeviceSynchronize();
}


void master(){

  thread model_thread(f1);
  thread tile_thread(f2);
  thread stream_thread(f3);

  model_thread.join();
  tile_thread.join();
  stream_thread.join();

  thread dl_thread(f4);
  dl_thread.join();

  printf("waiting for master thread to return\n");

  


}

void printCudaVersion(){
printf("somewhere here\n");
  int err = hipSetDevice(0);
  master();
  hipDeviceSynchronize();
  Mat image = imread("/home/joker/cmake-cuda-example-master/cat.jpg", -1);
  imshow("image", image);
  waitKey(0);
//  return 0;
}

/*
int main()
{
printCudaVersion();
return 0;

}
*/
